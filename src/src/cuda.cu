#include "hip/hip_runtime.h"
#include "cuda.cuh"

#define PRINT_VEC3(vec) printf("%.2f, %.2f, %.2f\n", vec.x, vec.y, vec.z);
#define PRINT_VEC4(vec) printf("%.2f, %.2f, %.2f, %.2f\n", vec.x, vec.y, vec.z, vec.w);

struct Ray;
struct HitPayload;

__device__ unsigned int toRGBA(glm::vec4& color)
{
	unsigned char r = color.r * 255.0f;
	unsigned char g = color.g * 255.0f;
	unsigned char b = color.b * 255.0f;
	unsigned char a = color.a * 255.0f;

	return (r << 24) | (g << 16) | (b << 8) | a;
}

__device__ HitPayload miss(const Ray& ray)
{
	HitPayload payload;
	payload.hitDistance = -1.0f;
	return payload;
}

__device__ HitPayload lightHit(const Ray& ray, int lightIndex)
{
	HitPayload payload;
	payload.hitDistance = 0.0f;
	payload.objectIndex = lightIndex;
	return payload;
}

__device__ HitPayload closestHit(const Ray& ray, int sphereIndex, float hitDistance, Scene scene)
{
	HitPayload payload;

	payload.hitDistance = hitDistance;
	payload.objectIndex = sphereIndex;

	glm::vec3 sphereCenter = scene.cudaSpherePositions[sphereIndex];

	payload.hitPoint = ray.origin + ray.direction * hitDistance;
	payload.normal = glm::normalize(payload.hitPoint - sphereCenter);

	return payload;
}

__device__ HitPayload traceRayFromPixel(const Ray& ray, Scene scene)
{
	int hitSphereIndex = -1;
	int hitLightIndex = -1;
	float hitDistance = FLT_MAX;

	for (int k = 0; k < scene.sphereCount; k++)
	{
		glm::vec3 origin = ray.origin - scene.cudaSpherePositions[k];
		glm::vec3 direction = ray.direction;

		float radius = scene.cudaSphereRadii[k];

		float a = glm::dot(direction, direction);
		float b = 2.0f * glm::dot(origin, direction);
		float c = glm::dot(origin, origin)
			- radius * radius;

		float delta = b * b - 4.0f * a * c;
		if (delta < 0)
			continue;

		float t = (-b - glm::sqrt(delta)) / (2.0f * a);

		if (t > 0 && t < hitDistance)
		{
			hitDistance = t;
			hitSphereIndex = k;
		}
	}

	for (int k = 0; k < scene.lightCount; k++)
	{
		if (!scene.cudaLightBools[k])
			continue;

		glm::vec3 origin = ray.origin - scene.cudaLightPositions[k];
		glm::vec3 direction = ray.direction;

		float a = glm::dot(direction, direction);
		float b = 2.0f * glm::dot(origin, direction);
		float c = glm::dot(origin, origin)
			- scene.lightRadius * scene.lightRadius;

		float delta = b * b - 4.0f * a * c;
		if (delta < 0)
			continue;

		float t = (-b - glm::sqrt(delta)) / (2.0f * a);

		if (t > 0 && t < hitDistance)
		{
			hitDistance = t;
			hitSphereIndex = -2;
			hitLightIndex = k;
		}

	}

	if (hitSphereIndex == -1)
		return miss(ray);

	if (hitSphereIndex == -2)
		return lightHit(ray, hitLightIndex);

	return closestHit(ray, hitSphereIndex, hitDistance, scene);
}

__device__ HitPayload traceRayFromHitpoint(Ray& ray, float diff, Scene scene)
{
	int hitSphereIndex = -1;
	float hitDistance = FLT_MAX;

	for (int k = 0; k < scene.sphereCount; k++)
	{
		glm::vec3 origin = ray.origin - scene.cudaSpherePositions[k];
		glm::vec3 direction = ray.direction;

		float radius = scene.cudaSphereRadii[k];

		float a = glm::dot(direction, direction);
		float b = 2.0f * glm::dot(origin, direction);
		float c = glm::dot(origin, origin)
			- radius * radius;

		float delta = b * b - 4.0f * a * c;
		if (delta < 0)
			continue;

		float t = (-b - glm::sqrt(delta)) / (2.0f * a);

		if (t > 0 && t < diff && t < hitDistance)
		{
			hitDistance = t;
			hitSphereIndex = k;
		}
	}

	if (hitSphereIndex == -1)
		return miss(ray);

	return closestHit(ray, hitSphereIndex, hitDistance, scene);
}

__device__ glm::vec4 phong(HitPayload payload, int lightIndex, Scene scene, glm::vec3 cameraPos, float d)
{
	glm::vec3 lightDir = glm::normalize(scene.cudaLightPositions[lightIndex] - payload.hitPoint);
	glm::vec3 lightColor = scene.cudaLightColors[lightIndex];
	float cosNL = glm::max(0.0f, glm::dot(lightDir, payload.normal));
	glm::vec3 reflectionVector = glm::reflect(-lightDir, payload.normal);
	glm::vec3 eyeVector = glm::normalize(cameraPos - payload.hitPoint);
	float cosVR = glm::max(0.0f, glm::dot(reflectionVector, eyeVector));

	glm::vec3 color =
		scene.params.kDiffuse * cosNL * lightColor +
		scene.params.kSpecular * glm::pow(cosVR, scene.params.kShininess) * lightColor;
	
	float attenuation = 1.0f / (1.0f + d * (scene.params.linearAtt + scene.params.quadraticAtt * d));
	color *= attenuation * scene.cudaSphereAlbedos[payload.objectIndex];

	return glm::vec4(color, 1.0f);
}

__device__ glm::vec4 rayGen(int i, int j, glm::vec3 origin,
	glm::vec3 direction, Scene scene, bool shadows)
{
	Ray ray;

	ray.origin = origin;
	ray.direction = direction;

	HitPayload payload = traceRayFromPixel(ray, scene);
	int idx = payload.objectIndex;

	// no sphere detected
	if (payload.hitDistance < 0)
		return glm::vec4(scene.params.skyColor, 1.0f);

	// light source hit
	if (payload.hitDistance == 0)
		return glm::vec4(scene.cudaLightColors[idx], 1.0f);

	glm::vec4 color = glm::vec4(scene.params.kAmbient * scene.params.ambientColor * scene.cudaSphereAlbedos[idx], 1.0f);

	// cast rays from hitpoint to light sources
	for (int lightIdx = 0; lightIdx < scene.lightCount; lightIdx++)
	{
		if (!scene.cudaLightBools[lightIdx])
			continue;

		float distanceToLight = glm::length(scene.cudaLightPositions[lightIdx] - payload.hitPoint);

		if (shadows)
		{
			Ray rayToLight;

			// cast ray a bit away from the sphere so that the ray doesn't hit it
			rayToLight.origin = payload.hitPoint + payload.normal * 1e-3f;
			rayToLight.direction = glm::normalize(scene.cudaLightPositions[lightIdx] - payload.hitPoint);

			HitPayload payloadToLight = traceRayFromHitpoint(rayToLight, distanceToLight, scene);

			// no sphere hit on path to light
			if (payloadToLight.hitDistance < 0)
				color += phong(payload, lightIdx, scene, origin, distanceToLight);
		}
		else
			color += phong(payload, lightIdx, scene, origin, distanceToLight);

	}

	return glm::clamp(color, 0.0f, 1.0f);
}

__device__ glm::vec3 getRayDirection(int i, int j, cudaArguments args)
{
	glm::vec2 coord{
		(float)j / args.width,
		(float)i / args.height
	};

	coord = coord * 2.0f - 1.0f;

	glm::vec4 target = args.inverseProjMatrix * glm::vec4(coord.x, coord.y, 1.0f, 1.0f);
	glm::vec3 rayDirection = glm::vec3(args.inverseViewMatrix * glm::vec4(glm::normalize(glm::vec3(target) / target.w), 0));

	return rayDirection;
}

__global__ void rayTrace(cudaArguments args)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= args.width || y >= args.height)
		return;

	int k = x + y * args.width;

	GLuint res = toRGBA(rayGen(y, x,
		args.rayOrigin, getRayDirection(y, x, args), 
		args.scene, args.shadows));

	args.cudaImage[k] = res;
}